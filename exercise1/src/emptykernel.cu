#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void emptyKernel()
{
    printf("empty kernel call\n");
}

int main()
{
    dim3 threadsPerBlock(1);
    dim3 blocksPerGrid(1);
    emptyKernel<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
    return 0;
}
