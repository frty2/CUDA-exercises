#include <iostream>
#include <glog/logging.h>

#include "copyprimitives.h"


void copyPrimitives(const primitives &objects)
{
    std::cout << "copying primitives" << std::endl;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipError_t error;
	triangle *deviceobjects;
	size_t sizeinbytes = objects.count*sizeof(triangle);
	
	error = hipMalloc(&deviceobjects, sizeinbytes);
	CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	hipEventRecord(start, 0);
	error = hipMemcpy(deviceobjects, objects.triangles, sizeinbytes, hipMemcpyHostToDevice);
	CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	std::cout << elapsedTime << " ms elapsed for copy operation" << std::endl;
}