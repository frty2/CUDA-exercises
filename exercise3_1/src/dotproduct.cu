#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include "hip/hip_vector_types.h"

#define itemcount 1024*1024

typedef float3 vector;

__global__ void dotproduct(vector *A, vector *B, float *C)
{
	int threadid = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
    if(threadid < itemcount){
    	C[threadid] = A[threadid].x*B[threadid].x + A[threadid].y*B[threadid].y + A[threadid].z*B[threadid].z;
    }
}

int main()
{
    size_t vecsize = itemcount*sizeof(vector);
    size_t fltsize = itemcount*sizeof(float);
    hipError_t error;
    vector *A;
    vector *B;
    float *C;

    hipHostAlloc(&A, vecsize, hipHostMallocDefault);
    hipHostAlloc(&B, vecsize, hipHostMallocDefault);
    hipHostAlloc(&C, fltsize, hipHostMallocDefault);

    if(A == NULL || B == NULL || C == NULL)
    {
        std::cout << "Not enough memory on host" << std::endl;
        return -1;
    }

    vector *deviceA;
    vector *deviceB;
    float *deviceC;
    hipMalloc(&deviceA, vecsize);
    hipMalloc(&deviceB, vecsize);
    hipMalloc(&deviceC, fltsize);

    if(deviceA == NULL || deviceB == NULL || deviceC == NULL)
    {
        std::cout << "Not enough memory on device" << std::endl;
        return -1;
    }

    for(int i = 0; i < itemcount; i++)
    {
        C[i] = 42;
        A[i].x = (rand() % 10000 + 1);
        A[i].y = (rand() % 10000 + 1);
        A[i].z = (rand() % 10000 + 1);
        B[i].x = (rand() % 10000 + 1);
        B[i].y = (rand() % 10000 + 1);
        B[i].z = (rand() % 10000 + 1);
    }

    error = hipMemcpy(deviceA, A, vecsize, hipMemcpyHostToDevice);
    if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(deviceB, B, vecsize, hipMemcpyHostToDevice);
    if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);

    hipEvent_t start, stop;
    float elapsedTime;

	int maxgrid = 65535;

    for(int i = 0; i <= 10; i++)
    {
        int threads = 1 << i;
        int blocks = (itemcount+threads-1)/threads;
		int gridx = maxgrid;
		int gridy = blocks / maxgrid + 1;
        std::cout << "Threads per block: " << threads << std::endl;
        std::cout << "Blocks per Grid: " << gridx << "x" << gridy << std::endl;
        dim3 threadsPerBlock(threads);
        dim3 blocksPerGrid(gridx, gridy);
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        dotproduct<<<blocksPerGrid, threadsPerBlock>>>(deviceA,deviceB,deviceC);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);

        hipEventDestroy(start);
        hipEventDestroy(stop);
        std::cout << elapsedTime << " ms elapsed for executing kernel" << std::endl;
        std::cout << std::endl;
    }


    hipDeviceSynchronize();

    error = hipMemcpy(C, deviceC, fltsize, hipMemcpyDeviceToHost);
    if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
}