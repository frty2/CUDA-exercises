#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include "hip/hip_vector_types.h"

#define itemcount 1024*1024

typedef float3 vector;

__global__ void dot(vector *A, vector *B, float *C)
{
	int threadid = threadIdx.x + blockIdx.x*blockDim.x;
	if(threadid < itemcount){
		C[threadid] = 97;//A[threadid].x*B[threadid].x + A[threadid].y*B[threadid].y + A[threadid].z*B[threadid].z;
	}
}

int main()
{
	size_t vecsize = itemcount*sizeof(vector);
	size_t fltsize = itemcount*sizeof(float);
	hipError_t error;
	vector *A;vector *B;float *C;
	
	hipHostAlloc(&A, vecsize, hipHostMallocDefault);
	hipHostAlloc(&B, vecsize, hipHostMallocDefault);
	hipHostAlloc(&C, fltsize, hipHostMallocDefault);
	
	if(A == NULL || B == NULL || C == NULL)
	{
		std::cout << "Not enough memory on host" << std::endl;
		return -1;
	}
	vector *deviceA;vector *deviceB;float *deviceC;
	hipMalloc(&deviceA, vecsize);
	hipMalloc(&deviceB, vecsize);
	hipMalloc(&deviceC, fltsize);
	
	if(deviceA == NULL || deviceB == NULL || deviceC == NULL)
	{
		std::cout << "Not enough memory on device" << std::endl;
		return -1;
	}
	
	for(int i = 0;i < itemcount;i++)
	{
		C[i] = 42;
		A[i].x = (rand() % 10000 + 1);
		A[i].y = (rand() % 10000 + 1);
		A[i].z = (rand() % 10000 + 1);
		B[i].x = (rand() % 10000 + 1);
		B[i].y = (rand() % 10000 + 1);
		B[i].z = (rand() % 10000 + 1);
	}
	
	error = hipMemcpy(deviceA, A, vecsize, hipMemcpyHostToDevice);
	if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(deviceB, B, vecsize, hipMemcpyHostToDevice);
	if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);
	
	
	dim3 threadsPerBlock(1024);
	dim3 blocksPerGrid(1024);
	dot<<<blocksPerGrid, threadsPerBlock>>>(deviceA,deviceB,deviceC);
	
	hipDeviceSynchronize();
	
	error = hipMemcpy(C, deviceC, fltsize, hipMemcpyDeviceToHost);
	if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);

	std::cout << C[0] << std::endl;
	std::cout << C[56] << std::endl;
	
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);
}