#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include "hip/hip_vector_types.h"

#define itemcount 1024*1024

typedef float3 vector;

__global__ void dotproduct(vector *A, vector *B, float *C)
{
	C[0] = 1337;
//	int threadid = threadIdx.x + blockIdx.x*blockDim.x;
	//if(threadid < itemcount){
	//	C[threadid] = 97;//A[threadid].x*B[threadid].x + A[threadid].y*B[threadid].y + A[threadid].z*B[threadid].z;
	//}
}

int main()
{
	size_t vecsize = itemcount*sizeof(vector);
	size_t fltsize = itemcount*sizeof(float);
	hipError_t error;
	vector *A;vector *B;float *C;
	
	hipHostAlloc(&A, vecsize, hipHostMallocDefault);
	hipHostAlloc(&B, vecsize, hipHostMallocDefault);
	hipHostAlloc(&C, fltsize, hipHostMallocDefault);
	
	if(A == NULL || B == NULL || C == NULL)
	{
		std::cout << "Not enough memory on host" << std::endl;
		return -1;
	}
	
	vector *deviceA;vector *deviceB;float *deviceC;
	hipMalloc(&deviceA, vecsize);
	hipMalloc(&deviceB, vecsize);
	hipMalloc(&deviceC, fltsize);
	
	if(deviceA == NULL || deviceB == NULL || deviceC == NULL)
	{
		std::cout << "Not enough memory on device" << std::endl;
		return -1;
	}
	
	for(int i = 0;i < itemcount;i++)
	{
		C[i] = 42;
		A[i].x = (rand() % 10000 + 1);
		A[i].y = (rand() % 10000 + 1);
		A[i].z = (rand() % 10000 + 1);
		B[i].x = (rand() % 10000 + 1);
		B[i].y = (rand() % 10000 + 1);
		B[i].z = (rand() % 10000 + 1);
	}
	
	error = hipMemcpy(deviceA, A, vecsize, hipMemcpyHostToDevice);
	if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(deviceB, B, vecsize, hipMemcpyHostToDevice);
	if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);
	
	hipEvent_t start, stop;
	float elapsedTime;
	
	
	for(int i = 0;i <= 10;i++){
		int threads = 1 << i;
		int blocks = (itemcount+threads-1)/threads;
		std::cout << "Threads per block: " << threads << std::endl;
		std::cout << "Blocks per Grid: " << blocks << std::endl;
		dim3 threadsPerBlock(threads);
		dim3 blocksPerGrid(blocks);
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		dotproduct<<<blocksPerGrid, threadsPerBlock>>>(deviceA,deviceB,deviceC);
		hipDeviceSynchronize();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);
		std::cout << elapsedTime << " ms elapsed for executing kernel" << std::endl;
		std::cout << std::endl;
	}
	
	
	hipDeviceSynchronize();
	
	error = hipMemcpy(C, deviceC, fltsize, hipMemcpyDeviceToHost);
	if(hipSuccess != error) std::cout << "Error: " << hipGetErrorString(error);
	
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);
}